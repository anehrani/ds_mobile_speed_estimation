/*

 */


#include <hip/hip_runtime.h>
#include <stdint.h>

__global__ void gpuYoloLayer_nc(
    const float* input, int* d_indexes, float* d_scores, float* d_boxes, int* d_classes, int* countData,
    const float scoreThreshold, const uint netWidth, const uint netHeight, const uint gridSizeX, const uint gridSizeY,
    const uint numOutputClasses, const uint numBBoxes, const float scaleXY, const float* anchors, const int* mask)
{
    uint x_id = blockIdx.x * blockDim.x + threadIdx.x;
    uint y_id = blockIdx.y * blockDim.y + threadIdx.y;
    uint z_id = blockIdx.z * blockDim.z + threadIdx.z;

    if (x_id >= gridSizeX || y_id >= gridSizeY || z_id >= numBBoxes)
        return;

    const int numGridCells = gridSizeX * gridSizeY;
    const int bbindex = y_id * gridSizeX + x_id;

    const float objectness
        = input[bbindex + numGridCells * (z_id * (5 + numOutputClasses) + 4)];

    if (objectness < scoreThreshold)
        return;

    int count = (int)atomicAdd(countData, 1);

    const float alpha = scaleXY;
    const float beta = -0.5 * (scaleXY - 1);

    float x
        = (input[bbindex + numGridCells * (z_id * (5 + numOutputClasses) + 0)]
          * alpha + beta + x_id) * netWidth / gridSizeX;

    float y
        = (input[bbindex + numGridCells * (z_id * (5 + numOutputClasses) + 1)]
          * alpha + beta + y_id) * netHeight / gridSizeY;

    float w
        = __powf(input[bbindex + numGridCells * (z_id * (5 + numOutputClasses) + 2)] * 2, 2)
          * anchors[mask[z_id] * 2];

    float h
        = __powf(input[bbindex + numGridCells * (z_id * (5 + numOutputClasses) + 3)] * 2, 2)
          * anchors[mask[z_id] * 2 + 1];

    float maxProb = 0.0f;
    int maxIndex = -1;

    for (uint i = 0; i < numOutputClasses; ++i)
    {
        float prob
            = input[bbindex + numGridCells * (z_id * (5 + numOutputClasses) + (5 + i))];

        if (prob > maxProb)
        {
            maxProb = prob;
            maxIndex = i;
        }
    }

    d_indexes[count] = count;
    d_scores[count] = objectness * maxProb + 1.f;
    d_boxes[count * 4 + 0] = x - 0.5 * w;
    d_boxes[count * 4 + 1] = y - 0.5 * h;
    d_boxes[count * 4 + 2] = x + 0.5 * w;
    d_boxes[count * 4 + 3] = y + 0.5 * h;
    d_classes[count] = maxIndex;
}

hipError_t cudaYoloLayer_nc(
    const void* input, void* d_indexes, void* d_scores, void* d_boxes, void* d_classes, void* countData,
    const uint& batchSize, uint64_t& inputSize, uint64_t& outputSize, const float& scoreThreshold, const uint& netWidth,
    const uint& netHeight, const uint& gridSizeX, const uint& gridSizeY, const uint& numOutputClasses, const uint& numBBoxes,
    const float& scaleXY, const void* anchors, const void* mask, hipStream_t stream);

hipError_t cudaYoloLayer_nc(
    const void* input, void* d_indexes, void* d_scores, void* d_boxes, void* d_classes, void* countData,
    const uint& batchSize, uint64_t& inputSize, uint64_t& outputSize, const float& scoreThreshold, const uint& netWidth,
    const uint& netHeight, const uint& gridSizeX, const uint& gridSizeY, const uint& numOutputClasses, const uint& numBBoxes,
    const float& scaleXY, const void* anchors, const void* mask, hipStream_t stream)
{
    dim3 threads_per_block(16, 16, 4);
    dim3 number_of_blocks((gridSizeX / threads_per_block.x) + 1,
                          (gridSizeY / threads_per_block.y) + 1,
                          (numBBoxes / threads_per_block.z) + 1);

    for (unsigned int batch = 0; batch < batchSize; ++batch)
    {
        gpuYoloLayer_nc<<<number_of_blocks, threads_per_block, 0, stream>>>(
            reinterpret_cast<const float*>(input) + (batch * inputSize),
            reinterpret_cast<int*>(d_indexes) + (batch * outputSize),
            reinterpret_cast<float*>(d_scores) + (batch * outputSize),
            reinterpret_cast<float*>(d_boxes) + (batch * 4 * outputSize),
            reinterpret_cast<int*>(d_classes) + (batch * outputSize), reinterpret_cast<int*>(countData) + (batch),
            scoreThreshold, netWidth, netHeight, gridSizeX, gridSizeY, numOutputClasses, numBBoxes, scaleXY,
            reinterpret_cast<const float*>(anchors), reinterpret_cast<const int*>(mask));
    }
    return hipGetLastError();
}
