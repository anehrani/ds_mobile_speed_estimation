/*

 */


#include <hip/hip_runtime.h>
#include <stdint.h>
#include <stdio.h>

__global__ void gpuYoloLayer_e(
    const float* cls, const float* reg, int* d_indexes, float* d_scores, float* d_boxes, int* d_classes, int* countData,
    const float scoreThreshold, const uint netWidth, const uint netHeight, const uint numOutputClasses,
    const uint64_t outputSize)
{
    uint x_id = blockIdx.x * blockDim.x + threadIdx.x;

    if (x_id >= outputSize)
        return;

    float maxProb = 0.0f;
    int maxIndex = -1;

    for (uint i = 0; i < numOutputClasses; ++i)
    {
        float prob
            = cls[x_id * numOutputClasses + i];

        if (prob > maxProb)
        {
            maxProb = prob;
            maxIndex = i;
        }
    }

    if (maxProb < scoreThreshold)
        return;

    int count = (int)atomicAdd(countData, 1);

    d_indexes[count] = count;
    d_scores[count] = maxProb + 1.f;
    d_boxes[count * 4 + 0] = reg[x_id * 4 + 0];
    d_boxes[count * 4 + 1] = reg[x_id * 4 + 1];
    d_boxes[count * 4 + 2] = reg[x_id * 4 + 2];
    d_boxes[count * 4 + 3] = reg[x_id * 4 + 3];
    d_classes[count] = maxIndex;
}

hipError_t cudaYoloLayer_e(
    const void* cls, const void* reg, void* d_indexes, void* d_scores, void* d_boxes, void* d_classes, void* countData,
    const uint& batchSize, uint64_t& outputSize, const float& scoreThreshold, const uint& netWidth, const uint& netHeight,
    const uint& numOutputClasses, hipStream_t stream);

hipError_t cudaYoloLayer_e(
    const void* cls, const void* reg, void* d_indexes, void* d_scores, void* d_boxes, void* d_classes, void* countData,
    const uint& batchSize, uint64_t& outputSize, const float& scoreThreshold, const uint& netWidth, const uint& netHeight,
    const uint& numOutputClasses, hipStream_t stream)
{
    int threads_per_block = 16;
    int number_of_blocks = 525;

    for (unsigned int batch = 0; batch < batchSize; ++batch)
    {
        gpuYoloLayer_e<<<number_of_blocks, threads_per_block, 0, stream>>>(
            reinterpret_cast<const float*>(cls) + (batch * numOutputClasses * outputSize),
            reinterpret_cast<const float*>(reg) + (batch * 4 * outputSize),
            reinterpret_cast<int*>(d_indexes) + (batch * outputSize),
            reinterpret_cast<float*>(d_scores) + (batch * outputSize),
            reinterpret_cast<float*>(d_boxes) + (batch * 4 * outputSize),
            reinterpret_cast<int*>(d_classes) + (batch * outputSize), reinterpret_cast<int*>(countData) + (batch),
            scoreThreshold, netWidth, netHeight, numOutputClasses, outputSize);
    }
    return hipGetLastError();
}
