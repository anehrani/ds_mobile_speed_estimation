#include "hip/hip_runtime.h"
/*

 */

#include <cub/device/device_radix_sort.cuh>

__global__ void sortOutput(
    int* d_indexes, float* d_scores, float* d_boxes, int* d_classes, float* bboxData, float* scoreData,
    const uint numOutputClasses, const int topk)
{
    uint x_id = blockIdx.x * blockDim.x + threadIdx.x;

    if (x_id >= topk)
        return;

    int index = d_indexes[x_id];
    int maxIndex = d_classes[index];
    bboxData[x_id * 4 + 0] = d_boxes[index * 4 + 0];
    bboxData[x_id * 4 + 1] = d_boxes[index * 4 + 1];
    bboxData[x_id * 4 + 2] = d_boxes[index * 4 + 2];
    bboxData[x_id * 4 + 3] = d_boxes[index * 4 + 3];
    scoreData[x_id * numOutputClasses + maxIndex] = d_scores[x_id] - 1.f;
}

hipError_t sortDetections(
    void* d_indexes, void* d_scores, void* d_boxes, void* d_classes, void* bboxData, void* scoreData, void* countData,
    const uint& batchSize, uint64_t& outputSize, uint& topK, const uint& numOutputClasses, hipStream_t stream);

hipError_t sortDetections(
    void* d_indexes, void* d_scores, void* d_boxes, void* d_classes, void* bboxData, void* scoreData, void* countData,
    const uint& batchSize, uint64_t& outputSize, uint& topK, const uint& numOutputClasses, hipStream_t stream)
{
    for (unsigned int batch = 0; batch < batchSize; ++batch)
    {
        int* _d_indexes = reinterpret_cast<int*>(d_indexes) + (batch * outputSize);
        float* _d_scores = reinterpret_cast<float*>(d_scores) + (batch * outputSize);

        int* _countData = reinterpret_cast<int*>(countData) + (batch);
        int count;
        hipMemcpy(&count, _countData, sizeof(int), hipMemcpyDeviceToHost);

        if (count == 0)
        {
            return hipGetLastError();
        }

        size_t begin_bit = 0;
        size_t end_bit = sizeof(float) * 8;

        float *d_keys_out = NULL;
        int *d_values_out = NULL;

        hipMalloc((void **)&d_keys_out, count * sizeof(float));
        hipMalloc((void **)&d_values_out, count * sizeof(int));

        void* d_temp_storage = NULL;
        size_t temp_storage_bytes = 0;

        hipcub::DeviceRadixSort::SortPairsDescending(d_temp_storage, temp_storage_bytes, _d_scores, d_keys_out, _d_indexes,
        d_values_out, count, begin_bit, end_bit);

        hipMalloc(&d_temp_storage, temp_storage_bytes);

        hipcub::DeviceRadixSort::SortPairsDescending(d_temp_storage, temp_storage_bytes, _d_scores, d_keys_out, _d_indexes,
        d_values_out, count, begin_bit, end_bit);

        hipMemcpy(_d_scores, d_keys_out, count * sizeof(float), hipMemcpyDeviceToDevice);
        hipMemcpy(_d_indexes, d_values_out, count * sizeof(int), hipMemcpyDeviceToDevice);

        int _topK = count < topK ? count : topK;

        int threads_per_block = 16;
        int number_of_blocks = 0;

        if (_topK % 2 == 0 && _topK >= threads_per_block)
            number_of_blocks = _topK / threads_per_block;
        else
            number_of_blocks = (_topK / threads_per_block) + 1;

        sortOutput<<<number_of_blocks, threads_per_block, 0, stream>>>(
            _d_indexes, _d_scores, reinterpret_cast<float*>(d_boxes) + (batch * 4 * outputSize),
            reinterpret_cast<int*>(d_classes) + (batch * outputSize),
            reinterpret_cast<float*>(bboxData) + (batch * topK * 4),
            reinterpret_cast<float*>(scoreData) + (batch * topK * numOutputClasses), numOutputClasses, _topK);

        hipFree(d_keys_out);
        hipFree(d_values_out);
        hipFree(d_temp_storage);
    }
    return hipGetLastError();
}
